#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "d_colorToGreyscale.h"
#include "CHECK.h"

#define CHANNELS 3
#define BLOCKSIZE 32
__global__ void d_colorToGreyscaleKernel(unsigned char *, unsigned char *,
                                         int, int);
/*
   d_colorToGreyscale
   Performs the greyscale of an image on the GPU.
   Pout array is filled with the greyscale of each pixel.
   Pin array contains the color pixels.
   width and height are the dimensions of the image.
*/
float d_colorToGreyscale(unsigned char * Pout, unsigned char * Pin,
                        int width, int height)
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    //Your work goes here and in the kernel below
    int size = width * height * sizeof(unsigned char);
    unsigned char * PoutCuda;
    unsigned char * PinCuda;

    CHECK(hipMalloc((void **) &PoutCuda, size));
    CHECK(hipMalloc((void **) &PinCuda, size*CHANNELS));

    CHECK(hipMemcpy(PinCuda, Pin, size*CHANNELS, hipMemcpyHostToDevice));

    dim3 grid(ceil(width/BLOCKSIZE)+1, ceil(height/BLOCKSIZE)+1, 1);
    dim3 block(BLOCKSIZE, BLOCKSIZE, 1);
    
    d_colorToGreyscaleKernel<<<grid, block>>>(PinCuda, PoutCuda, width, height);

    CHECK(hipMemcpy(Pout, PoutCuda, size, hipMemcpyDeviceToHost));
    CHECK(hipFree(PoutCuda)); 
    CHECK(hipFree(PinCuda));
    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

/*
   d_colorToGreyscaleKernel
   Kernel code executed by each thread on its own data when the kernel is
   launched.
   Pout array is filled with the greyscale of each pixel (one element per thread).
   Pin array contains the color pixels.
   width and height are the dimensions of the image.
*/
__global__
void d_colorToGreyscaleKernel(unsigned char * Pin, unsigned char * Pout,
                              int width, int height)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if( col < width && row < height ){
		int flat = row*width + col;
		int offset = CHANNELS * flat;
		unsigned char r = Pin[offset];
		unsigned char g = Pin[offset+1];
		unsigned char b = Pin[offset+2];
		Pout[flat] = 0.21f*r + 0.71f*g + 0.07f*b;
	}
}
